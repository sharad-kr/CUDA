
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to print "Hello, World!" from each thread
__global__ void helloFromGPU() {
    // Calculate the thread's unique ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Print "Hello, World!" from each thread
    printf("Hello, World! from thread %d\n", tid);
}

int main() {
    // Define grid and block dimensions
    int numBlocks = 1;    // Number of blocks
    int blockSize = 10;   // Threads per block

    // Launch the CUDA kernel with specified grid and block dimensions
    helloFromGPU<<<numBlocks, blockSize>>>();

    // Synchronize threads after kernel execution
    hipDeviceSynchronize();

    // Check for any errors during the kernel launch
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    return 0;
}

